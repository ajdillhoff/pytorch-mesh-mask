#include "hip/hip_runtime.h"
#include <cstdio>
#include <cfloat>
#include <cmath>
#include <cassert>
#include <vector>

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


typedef long long int64;

__device__ constexpr float kMinimumTriangleArea() { return 1e-13; }
__device__ constexpr float kDegenerateBarycentricCoordinatesCutoff() { return 0.9f; }

#define gpuErrorcheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUAssert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) {
            exit(code);
        }
    }
}

__device__ int ClampedIntegerMin(float a, float b, float c, int low, int high) {
    return min(
            max(float2int(floor(min(min(a, b), c))), low),
            high);
}

__device__ int ClampedIntegerMax(float a, float b, float c, int low, int high) {
    return min(
            max(float2int(ceil(max(max(a, b), c))), low),
            high);
}

__device__ int FixedPoint(float f) { return float2int(f * (1 << 16)); }

__device__ bool IsCCW(int v0x, int v0y, int v1x, int v1y, int px, int py) {
    int ex = v1x - v0x;
    int ey = v1y - v0y;
    int x = px - v0x;
    int y = py - v0y;
    int64 ex_y = int64{ex} * int64{y};
    int64 ey_x = int64{ey} * int64{x};
    return ex_y >= ey_x;
}

__device__ bool PixelIsInsideTriangle(int v0x, int v0y, int v1x, int v1y,
        int v2x, int v2y, int px, int py) {
    return (IsCCW(v0x, v0y, v1x, v1y, px, py) &&
            IsCCW(v1x, v1y, v2x, v2y, px, py) &&
            IsCCW(v2x, v2y, v0x, v0y, px, py)) ||
        (IsCCW(v1x, v1y, v0x, v0y, px, py) &&
         IsCCW(v2x, v2y, v1x, v1y, px, py) &&
         IsCCW(v0x, v0y, v2x, v2y, px, py));
}

__global__ void compute_mesh_mask_cuda_forward(
        float* vertices,
        int* triangles,
        float* z_buffer,
        int *mesh_mask,
        int triangle_count,
        int image_width,
        int image_height,
        int batch_size,
        int num_vertices) {

    const int triangle_id = threadIdx.x + blockIdx.x * blockDim.x;

    if (triangle_id >= triangle_count) { return; }

    const float half_image_width = 0.5 * image_width;
    const float half_image_height = 0.5 * image_height;

    for (int batch_id = 0; batch_id < batch_size; batch_id++) {

        const int v0_x_id = (batch_id * num_vertices * 3) + 3 * triangles[3 * triangle_id];
        const int v1_x_id = (batch_id * num_vertices * 3) + 3 * triangles[3 * triangle_id + 1];
        const int v2_x_id = (batch_id * num_vertices * 3) + 3 * triangles[3 * triangle_id + 2];

        // Convert NDC vertex positions to viewport coordinates.
        const float v0x = (vertices[v0_x_id] + 1.0) * half_image_width;
        const float v0y = (vertices[v0_x_id + 1] + 1.0) * half_image_height;
        const float v0z = vertices[v0_x_id + 2];
        const float v1x = (vertices[v1_x_id] + 1.0) * half_image_width;
        const float v1y = (vertices[v1_x_id + 1] + 1.0) * half_image_height;
        const float v1z = vertices[v1_x_id + 2];
        const float v2x = (vertices[v2_x_id] + 1.0) * half_image_width;
        const float v2y = (vertices[v2_x_id + 1] + 1.0) * half_image_height;
        const float v2z = vertices[v2_x_id + 2];

        // Compute the normal
        const float ux = v1x - v0x;
        const float uy = v1y - v0y;
        const float vx = v2x - v0x;
        const float vy = v2y - v0y;
        const float nz = ux * vy - uy * vx;

        if (nz <= 0) { continue; }

        // Find the triangle bounding box enlarged to the nearest integer and
        // clamped to the image boundaries.
        const int left = ClampedIntegerMin(v0x, v1x, v2x, 0, image_width);
        const int right = ClampedIntegerMax(v0x, v1x, v2x, 0, image_width);
        const int bottom = ClampedIntegerMin(v0y, v1y, v2y, 0, image_height);
        const int top = ClampedIntegerMax(v0y, v1y, v2y, 0, image_height);

        // Convert coordinates to fixed-point to make triangle intersection
        // testing consistent and prevent cracks.
        const int fv0x = FixedPoint(v0x);
        const int fv0y = FixedPoint(v0y);
        const int fv1x = FixedPoint(v1x);
        const int fv1y = FixedPoint(v1y);
        const int fv2x = FixedPoint(v2x);
        const int fv2y = FixedPoint(v2y);

        for (int i = bottom; i < top; i++) {
            for (int j = left; j < right; j++) {
                const float px = j + 0.5;
                const float py = i + 0.5;

                if (!PixelIsInsideTriangle(fv0x, fv0y, fv1x, fv1y, fv2x, fv2y,
                            FixedPoint(px), FixedPoint(py))) {
                    continue;
                }

                const int pixel_idx = (batch_id * image_width * image_height) + i * image_width + j;

                // Compute twice the area of two barycentric triangles, as well
                // as the triangle they sit in. The barycentric is the ratio of
                // the triangle areas, so the factor of two does not change the
                // result.
                const float twice_triangle_area =
                    (v2x - v0x) * (v1y - v0y) - (v2y - v0y) * (v1x - v0x);
                const float b0 = ((px - v1x) * (v2y - v1y) - (py - v1y) *
                        (v2x - v1x)) / twice_triangle_area;
                const float b1 = ((px - v2x) * (v0y - v2y) - (py - v2y) *
                        (v0x - v2x)) / twice_triangle_area;

                // The three upper triangle partition the lower triangle, so we
                // can compute the third barycentric coordinate using the other
                // two.
                const float b2 = 1.0f - b0 - b1;

                const float z = b0 * v0z + b1 * v1z + b2 * v2z;

                // Skip the pixel if it is farther than the current z-buffer
                // pixel or beyond the near or far clipping plane.
                if (z < -1.0 || z > 1.0 || z > z_buffer[pixel_idx]) {
                    // Atomic operation resolves race conditions for self-occlusion.
                    atomicCAS(&mesh_mask[v0_x_id / 3], 1, 0);
                    atomicCAS(&mesh_mask[v1_x_id / 3], 1, 0);
                    atomicCAS(&mesh_mask[v2_x_id / 3], 1, 0);
                    continue;
                }

                z_buffer[pixel_idx] = z;
                // Atomic operation resolves race conditions for self-occlusion.
                atomicCAS(&mesh_mask[v0_x_id / 3], 0, 1);
                atomicCAS(&mesh_mask[v1_x_id / 3], 0, 1);
                atomicCAS(&mesh_mask[v2_x_id / 3], 0, 1);
            }
        }
    }
}

std::vector<torch::Tensor> compute_mesh_mask_forward(
        torch::Tensor vertices,
        torch::Tensor triangles,
        int triangle_count,
        int image_width,
        int image_height) {
    const int batch_size = vertices.size(0);
    const int num_vertices = vertices.size(1);
    auto options = torch::TensorOptions()
        .dtype(torch::kInt32)
        .layout(torch::kStrided)
        .device(torch::kCUDA, 0)
        .requires_grad(false);
    auto z_buffer = torch::ones({batch_size, image_height * image_width}, vertices.type());
    auto mesh_mask = torch::zeros({batch_size, num_vertices}, options);

    int minGridSize;
    int blockSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, compute_mesh_mask_cuda_forward, 0, triangle_count);
    int gridSize = (triangle_count + blockSize - 1) / blockSize;

    dim3 block;
    block.x = blockSize;
    dim3 grid;
    grid.x = gridSize;

    compute_mesh_mask_cuda_forward<<<grid, block>>>(
            vertices.data<float>(),
            triangles.data<int>(),
            z_buffer.data<float>(),
            mesh_mask.data<int>(),
            triangle_count,
            image_width,
            image_height,
            batch_size,
            vertices.size(1));

    hipDeviceSynchronize();

    gpuErrorcheck(hipPeekAtLastError());

    return {mesh_mask.to(torch::kUInt8)};
}
